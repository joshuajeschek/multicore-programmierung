
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>
#define N 160000

void initVector(int *vec, int len, int min, int max) {
  for (int i = 0; i < len; i++) {
    vec[i] = min + rand() % (max - min);
  }
}
void printVector(int *vec, int len) {
  printf("[");
  for (int i = 0; i < len - 1; i++) {
    printf("%d,", vec[i]);
  }
  printf("%d]\n", vec[len - 1]);
}

__global__ void vecadd(int *a, int *b, int *c) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int i;
  for (i=tid; i<N; i += blockDim.x * gridDim.x)
    c[i] = a[i] + b[i];
}

int main(void) {
  srand(time(NULL));
  int a[N], b[N], c[N];
  int *ad, *bd, *cd;
  hipMalloc ((void **)&ad, N * sizeof(int));
  hipMalloc ((void **)&bd, N * sizeof(int));
  hipMalloc ((void **)&cd, N * sizeof(int));
  initVector(a, N, 1, 5);
  printVector(a, N);
  initVector(b, N, 1, 5);
  printVector(b, N);
  hipMemcpy(ad, a, N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(bd, b, N*sizeof(int), hipMemcpyHostToDevice);
  vecadd<<<100,160>>>(ad, bd, cd);
  hipMemcpy(c, cd, N*sizeof(int), hipMemcpyDeviceToHost);
  printVector(c, N);
  hipFree(ad); hipFree(bd); hipFree(cd);
  return 0;
}
